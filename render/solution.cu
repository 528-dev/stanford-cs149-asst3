#include "hip/hip_runtime.h"
namespace Solution3 {

constexpr int BLOCK_DIM = 16;
constexpr int BLOCK_SIZE = BLOCK_DIM * BLOCK_DIM;

#define SCAN_BLOCK_DIM BLOCK_SIZE
#include "exclusiveScan.cu_inl"

__global__ void kernelRenderCircles() {
  __shared__ uint circleIsInBox[BLOCK_SIZE];
  __shared__ uint circleIndex[BLOCK_SIZE];
  __shared__ uint scratch[2 * BLOCK_SIZE];
  __shared__ int inBoxCircles[BLOCK_SIZE];

  int boxL = blockIdx.x * BLOCK_DIM;
  int boxB = blockIdx.y * BLOCK_DIM;
  int boxR = min(boxL + BLOCK_DIM, cuConstRendererParams.imageWidth);
  int boxT = min(boxB + BLOCK_DIM, cuConstRendererParams.imageHeight);
  float boxLNorm = boxL * cuConstRendererParams.invWidth;
  float boxRNorm = boxR * cuConstRendererParams.invWidth;
  float boxTNorm = boxT * cuConstRendererParams.invHeight;
  float boxBNorm = boxB * cuConstRendererParams.invHeight;

  int index = threadIdx.y * BLOCK_DIM + threadIdx.x;
  int pixelX = boxL + threadIdx.x;
  int pixelY = boxB + threadIdx.y;
  int pixelId = pixelY * cuConstRendererParams.imageWidth + pixelX;

  for (int i = 0; i < cuConstRendererParams.numCircles; i += BLOCK_SIZE) {
    int circleId = i + index;
    if (circleId < cuConstRendererParams.numCircles) {
      float3 p = *reinterpret_cast<float3 *>(
          &cuConstRendererParams.position[3 * circleId]);
      circleIsInBox[index] =
          circleInBox(p.x, p.y, cuConstRendererParams.radius[circleId],
                      boxLNorm, boxRNorm, boxTNorm, boxBNorm);
    } else {
      circleIsInBox[index] = 0;
    }
    __syncthreads();

    sharedMemExclusiveScan(index, circleIsInBox, circleIndex, scratch,
                           BLOCK_SIZE);
    if (circleIsInBox[index]) {
      inBoxCircles[circleIndex[index]] = circleId;
    }
    __syncthreads();

    int numCirclesInBox =
        circleIndex[BLOCK_SIZE - 1] + circleIsInBox[BLOCK_SIZE - 1];
    __syncthreads();

    if (pixelX < boxR && pixelY < boxT) {
      float4 *imgPtr = reinterpret_cast<float4 *>(
          &cuConstRendererParams.imageData[4 * pixelId]);
      for (int j = 0; j < numCirclesInBox; j++) {
        circleId = inBoxCircles[j];
        shadePixel(
            circleId,
            make_float2((pixelX + 0.5) * cuConstRendererParams.invWidth,
                        (pixelY + 0.5) * cuConstRendererParams.invHeight),
            *reinterpret_cast<float3 *>(
                &cuConstRendererParams.position[3 * circleId]),
            imgPtr);
      }
    }
  }
}

void renderCircles(int width, int height) {
  kernelRenderCircles<<<dim3((width + BLOCK_DIM - 1) / BLOCK_DIM,
                             (height + BLOCK_DIM - 1) / BLOCK_DIM),
                        dim3(BLOCK_DIM, BLOCK_DIM)>>>();
  cudaCheckError(hipDeviceSynchronize());
}
} // namespace Solution3
